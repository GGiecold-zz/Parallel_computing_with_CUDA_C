#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../Common/errors.h"


#define N 100


__global__ void add(int *x, int *y, int *z)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;

  while (id < N) {
    z[id] = x[id] + y[id];
    id += blockDim.x * gridDim.x;
  }
}


int main(void)
{
  int x[N], *device_x;
  HANDLE_ERROR(hipMalloc((void **) &device_x, sizeof(int) * N));

  int y[N], *device_y;
  HANDLE_ERROR(hipMalloc((void **) &device_y, sizeof(int) * N));

  int z[N], *device_z;
  HANDLE_ERROR(hipMalloc((void **) &device_z, sizeof(int) * N));

  for (int i = 0; i < N; i++) {
    x[i] = i;
    y[i] = -i;
  }

  HANDLE_ERROR(hipMemcpy(device_x, x, sizeof(int) * N,
    hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(device_y, y, sizeof(int) * N,
    hipMemcpyHostToDevice));

  add<<<128, 128>>>(device_x, device_y, device_z);

  HANDLE_ERROR(hipMemcpy(z, device_z, sizeof(int) * N,
    hipMemcpyDeviceToHost));

  int fail_count = 0;  
  for (int i = 0; i < N; i++)
    if (z[i] != x[i] + y[i]) {
      printf("Error: %d + %d != %d\n", x[i], y[i], z[i]);
      fail_count += 1;
    }

  if (fail_count)
    printf("Encountered %d errors\n", fail_count);

  hipFree(device_x);
  hipFree(device_y);
  hipFree(device_z);

  return 0;
}

