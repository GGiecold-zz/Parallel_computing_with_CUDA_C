#include "hip/hip_runtime.h"
/* Illustrates task-parallelism and asynchronous kernel launch and
   memory copies using two CUDA streams, i.e. two independent queues
   of operations scheduled for execution on the GPU.
   Improved performance is achieved using the overlap capacity offered
   by most current GPU devices of doing kernel execution with memory
   copies between device and host in parallel. This is achieved through
   the GPU hardware having separate engines for those two executions.

   For more information on the mock chunking performed by the code
   herewith, see task_parallelism_with_single_CUDA_stream.cu,
   part of the same directory.
*/


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "../Common/errors.h"


#define BLOCK_SIZE ((long) 1024 * 1024)
#define SIZE ((long) 50 * BLOCK_SIZE)


__global__ void kernel(int *, int *, int *);


int main(void)
{
  srand(time(NULL));

  hipEvent_t start, stop;
  float compute_time;

  hipDeviceProp_t properties;
  int device_id;

  HANDLE_ERROR(hipGetDevice(&device_id));
  HANDLE_ERROR(hipGetDeviceProperties(&properties, device_id));
  
  if (false == (bool) properties.deviceOverlap) {
    printf("The GPU device cannot handle overlaps and therefore no "
           "performance improvement will be achieved via CUDA streams. "
	   "Exiting.\n");

    return 1;
  }

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  hipStream_t stream_0, stream_1;
  HANDLE_ERROR(hipStreamCreate(&stream_0));
  HANDLE_ERROR(hipStreamCreate(&stream_1));

  // We allocate memory for host arrays and corresponding GPU buffers
  // associated with stream_0 and stream_1:
  
  int *x, *device_x_0, *device_x_1;
  HANDLE_ERROR(hipHostAlloc((void **) &x, sizeof(int) * SIZE,
    hipHostMallocDefault));
  HANDLE_ERROR(hipMalloc((void **) &device_x_0,
    sizeof(int) * BLOCK_SIZE));
  HANDLE_ERROR(hipMalloc((void **) &device_x_1,
    sizeof(int) * BLOCK_SIZE));

  int *y, *device_y_0, *device_y_1;
  HANDLE_ERROR(hipHostAlloc((void **) &y, sizeof(int) * SIZE,
    hipHostMallocDefault));
  HANDLE_ERROR(hipMalloc((void **) &device_y_0,
    sizeof(int) * BLOCK_SIZE));
  HANDLE_ERROR(hipMalloc((void **) &device_y_1,
    sizeof(int) * BLOCK_SIZE));

  int *z, *device_z_0, *device_z_1;
  HANDLE_ERROR(hipHostAlloc((void **) &z, sizeof(int) * SIZE,
    hipHostMallocDefault));
  HANDLE_ERROR(hipMalloc((void **) &device_z_0,
    sizeof(int) * BLOCK_SIZE));
  HANDLE_ERROR(hipMalloc((void **) &device_z_1,
    sizeof(int) * BLOCK_SIZE));

  for (int i = 0; i < SIZE; i++) {
    x[i] = rand();
    y[i] = rand();
  }

  // Naively, we would copy chunks of the pinned host memory to device,
  // launch the kernel, then copy the output array in chunk
  // from device to host.
  // Those steps would be performed asynchronously by alternating with
  // two streams.
  // However, scheduling all of a given stream's operations at once
  // is often inefficient, with some copies and kernel
  // execution being blocked by others and waiting idly.
  // To bypass the performance bottleneck that would result,
  // it turns out to be more efficient to interleave
  // the copy and kernel execution tasks across streams,
  // as is done in the following lines of code:
  
  for (int i = 0; i < SIZE; i += 2 * BLOCK_SIZE) {
    HANDLE_ERROR(hipMemcpyAsync(device_x_0, x + i, sizeof(int) * BLOCK_SIZE,
      hipMemcpyHostToDevice, stream_0));
    HANDLE_ERROR(hipMemcpyAsync(device_x_1, x + i + BLOCK_SIZE,
      sizeof(int) * BLOCK_SIZE, hipMemcpyHostToDevice, stream_1));
      
    HANDLE_ERROR(hipMemcpyAsync(device_y_0, y + i, sizeof(int) * BLOCK_SIZE,
      hipMemcpyHostToDevice, stream_0));
    HANDLE_ERROR(hipMemcpyAsync(device_y_1, y + i + BLOCK_SIZE,
      sizeof(int) * BLOCK_SIZE, hipMemcpyHostToDevice, stream_1));

    kernel<<<BLOCK_SIZE / 256, 256, 0, stream_0>>>(
      device_x_0, device_y_0, device_z_0);
    kernel<<<BLOCK_SIZE / 256, 256, 0, stream_1>>>(
      device_x_1, device_y_1, device_z_1);

    HANDLE_ERROR(hipMemcpyAsync(z + i, device_z_0, sizeof(int) * BLOCK_SIZE,
      hipMemcpyDeviceToHost, stream_0));
    HANDLE_ERROR(hipMemcpyAsync(z + i + BLOCK_SIZE, device_z_1,
      sizeof(int) * BLOCK_SIZE, hipMemcpyDeviceToHost, stream_1));
  }
  // Makes sure the GPU streams are done by synchronizing with the CPU:
  HANDLE_ERROR(hipStreamSynchronize(stream_0));
  HANDLE_ERROR(hipStreamSynchronize(stream_1));

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&compute_time, start, stop));
  printf("Compute time: %.1f ms\n", compute_time);

  // Lots of cleanup:
  
  HANDLE_ERROR(hipStreamDestroy(stream_0));
  HANDLE_ERROR(hipStreamDestroy(stream_1));

  HANDLE_ERROR(hipHostFree(x));
  HANDLE_ERROR(hipFree(device_x_0));
  HANDLE_ERROR(hipFree(device_x_1));
  
  HANDLE_ERROR(hipHostFree(y));
  HANDLE_ERROR(hipFree(device_y_0));
  HANDLE_ERROR(hipFree(device_y_1));
  
  HANDLE_ERROR(hipHostFree(z));
  HANDLE_ERROR(hipFree(device_z_0));
  HANDLE_ERROR(hipFree(device_z_1));

  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  return 0;
}


__global__ void kernel(int *x, int *y, int *z)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < SIZE) {
    int id_1 = (id + 1) % 256;
    int id_2 = (id + 2) % 256;

    float sum_x = (x[id] + x[id_1] + x[id_2]) / 3.0f;
    float sum_y = (y[id] + y[id_1] + y[id_2]) / 3.0f;

    z[id] = (sum_x + sum_y) / 2.0f;
  }
}
