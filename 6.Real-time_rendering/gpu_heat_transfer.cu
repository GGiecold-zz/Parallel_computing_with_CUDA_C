#include "hip/hip_runtime.h"
/* A modification of our heat transfer simulation
   (see 5_About_texture_memory). Graphics interoperations enable us to
   achieve lower latency by getting rid of the repeated copies of a data buffer
   between the CPU and GPU for display that were part and parcel of our
   former implementation of that simulation (in other words, every frame
   computed on GPU had to be copied to the host before display, quite a waste).

   By creating and synchronizing CUDA events we can compare those two versions
   and indeed measure a significant gain in performance for the one at hand
   (typically 5.7 ms versus 4.3 ms per frame, using a single NVIDIA GTX 1070).

   As usual, compile through a call to

     $nvcc gpu_heat_transfer.cu -o gpu_heat_transfer.out -lGL -lGLU -lglut
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../Common/errors.h"
#include "../Common/gpu_animate.h"
#include "../Common/lib.h"


#define DIM 1024
#define MAX_TEMPERATURE 1.0f
#define MIN_TEMPERATURE 0.0001f
#define SPEED 0.2f
#define TIME_STEPS_PER_FRAME 50


struct Data {
  hipEvent_t start, stop;
  float compute_time;

  float count_frames;

  float *device_heaters;
  float *device_input;
  float *device_output;
};


void animate_display(uchar4 *, Data *, int);
void animate_exit(Data *);
__global__ void maintain_heaters(float *);
__global__ void temperature_update(float *, bool);


texture<float> texture_heaters;
texture<float> texture_input;
texture<float> texture_output;


int main(void)
{
  Data data;
  
  GPUAnimateBitmap bitmap(DIM, DIM, &data);

  data.compute_time = 0.0f;
  data.count_frames = 0;

  HANDLE_ERROR(hipEventCreate(&data.start));
  HANDLE_ERROR(hipEventCreate(&data.stop));

  HANDLE_ERROR(hipMalloc((void **) &data.device_heaters, bitmap.size()));
  HANDLE_ERROR(hipMalloc((void **) &data.device_input, bitmap.size()));
  HANDLE_ERROR(hipMalloc((void **) &data.device_output, bitmap.size()));

  HANDLE_ERROR(hipBindTexture(NULL, texture_heaters,
    data.device_heaters, bitmap.size()));
  HANDLE_ERROR(hipBindTexture(NULL, texture_input,
    data.device_input, bitmap.size()));
  HANDLE_ERROR(hipBindTexture(NULL, texture_output,
    data.device_output, bitmap.size()));

  float *heaters_grid = new float[bitmap.size()];
  for (int i = 0; i < DIM * DIM; i++) {
    heaters_grid[i] = 0.0f;

    int x = i % DIM;
    int y = i / DIM;

    if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
      heaters_grid[i] = MAX_TEMPERATURE;

    heaters_grid[100 + 100 * DIM] = (MIN_TEMPERATURE + MAX_TEMPERATURE) / 2;
    heaters_grid[100 + 700 * DIM] = MIN_TEMPERATURE;
    heaters_grid[300 + 300 * DIM] = MIN_TEMPERATURE;
    heaters_grid[700 + 200 * DIM] = MIN_TEMPERATURE;

    for (int k = 800; k < 900; k++) {
      for (int j = 400; j < 500; j++)
        heaters_grid[j + k * DIM] = MIN_TEMPERATURE;
    }
  }

  HANDLE_ERROR(hipMemcpy(data.device_heaters, heaters_grid,
    bitmap.size(), hipMemcpyHostToDevice));

  for (int j = 800; j < DIM; j++)
    for (int i = 0; i < 200; i++)
      heaters_grid[i + j * DIM] = MAX_TEMPERATURE;

  HANDLE_ERROR(hipMemcpy(data.device_heaters, heaters_grid,
    bitmap.size(), hipMemcpyHostToDevice));

  delete [] heaters_grid;

  bitmap.animate((void (*)(uchar4 *, void *, int)) animate_display,
    (void (*)(void *)) animate_exit);

  return 0;
}


void animate_display(uchar4 *ptr, Data *data, int clock)
{
  float compute_time;

  HANDLE_ERROR(hipEventRecord(data->start, 0));

  dim3 blocks_per_grid(DIM / 16, DIM / 16);
  dim3 threads_per_block(16, 16);
  volatile bool flag_IO = true;

  for (int i = 0; i < TIME_STEPS_PER_FRAME; i++) {
    float *input_ptr, *output_ptr;

    if (flag_IO == true) {
      input_ptr = data->device_input;
      output_ptr = data->device_output;
    } else {
      input_ptr = data->device_output;
      output_ptr = data->device_input;
    }

    maintain_heaters<<<blocks_per_grid, threads_per_block>>>(
      input_ptr);
    temperature_update<<<blocks_per_grid, threads_per_block>>>(
      output_ptr, flag_IO);

    flag_IO = !flag_IO;
  }

  float_to_color<<<blocks_per_grid, threads_per_block>>>(
    data->device_input, ptr);

  HANDLE_ERROR(hipEventRecord(data->stop, 0));
  HANDLE_ERROR(hipEventSynchronize(data->stop));
  HANDLE_ERROR(hipEventElapsedTime(&compute_time,
    data->start, data->stop));
  data->compute_time += compute_time;

  ++data->count_frames;

  printf("Mean compute time per frame: %.1f ms\n",
    data->compute_time / data->count_frames);
}


void animate_exit(Data *data)
{
  HANDLE_ERROR(hipUnbindTexture(texture_heaters));
  HANDLE_ERROR(hipUnbindTexture(texture_input));
  HANDLE_ERROR(hipUnbindTexture(texture_output));

  HANDLE_ERROR(hipFree(data->device_heaters));
  HANDLE_ERROR(hipFree(data->device_input));
  HANDLE_ERROR(hipFree(data->device_output));
  
  HANDLE_ERROR(hipEventDestroy(data->start));
  HANDLE_ERROR(hipEventDestroy(data->stop));
}


__global__ void maintain_heaters(float *ptr)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int current_idx = x + y * blockDim.x * gridDim.x;

  float heat_source = tex1Dfetch(texture_heaters, current_idx);
  if (heat_source != 0)
    ptr[current_idx] = heat_source;
}


__global__ void temperature_update(float *ptr, bool flag_IO)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int current_idx = x + y * blockDim.x * gridDim.x;

  int left_idx = current_idx - 1;
  if (x == 0)
    ++left_idx;
  int right_idx = current_idx + 1;
  if (x == DIM - 1)
    --right_idx;
  int upper_idx = current_idx - DIM;
  if (y == 0)
    upper_idx += DIM;
  int lower_idx = current_idx + DIM;
  if (y == DIM - 1)
    lower_idx -= DIM;

  float current, left, right, upper, lower;

  if (flag_IO) {
    current = tex1Dfetch(texture_input, current_idx);
    left = tex1Dfetch(texture_input, left_idx);
    right = tex1Dfetch(texture_input, right_idx);
    upper = tex1Dfetch(texture_input, upper_idx);
    lower = tex1Dfetch(texture_input, lower_idx);
  } else {
    current = tex1Dfetch(texture_output, current_idx);
    left = tex1Dfetch(texture_output, left_idx);
    right = tex1Dfetch(texture_output, right_idx);
    upper = tex1Dfetch(texture_output, upper_idx);
    lower = tex1Dfetch(texture_output, lower_idx);
  }

  float gradient = left + right + upper + lower - 4 * current;
  ptr[current_idx] = current + SPEED * gradient;
}
