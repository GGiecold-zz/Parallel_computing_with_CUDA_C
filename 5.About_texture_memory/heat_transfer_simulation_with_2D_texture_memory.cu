#include "hip/hip_runtime.h"
// $nvcc heat_transfer_simulation_with_2D_texture_memory.cu -o img.o -lGL -lGLU -glut


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../Common/animate.h"
#include "../Common/errors.h"
#include "../Common/lib.h"  // for the float_to_color kernel


#define DIM 1024
#define MAX_TEMPERATURE 1.0f
#define MIN_TEMPERATURE 0.0001f
#define SPEED 0.2f
#define TIME_STEPS_PER_FRAME 50


struct Data {
  hipEvent_t start, stop;
  float compute_time;

  AnimateBitmap *bitmap;
  unsigned char *device_bitmap;
  float count_frames;

  float *device_heaters;
  float *device_input;
  float *device_output;
};


void animate_display(Data *, int);
void animate_exit(Data *);
__global__ void maintain_heaters(float *);
__global__ void temperature_update(float *, bool);


/* Texture references must be declared at file scope.
   We will be using two-dimensional texture memory.
*/
texture<float, 2> texture_heaters;
texture<float, 2> texture_input;
texture<float, 2> texture_output;


int main(void)
{
  Data data;
  
  AnimateBitmap bitmap(DIM, DIM, &data);
  data.bitmap = &bitmap;
  
  data.compute_time = 0.0f;
  data.count_frames = 0;
  
  HANDLE_ERROR(hipEventCreate(&data.start));
  HANDLE_ERROR(hipEventCreate(&data.stop));

  HANDLE_ERROR(hipMalloc((void **) &data.device_bitmap, bitmap.size()));
  HANDLE_ERROR(hipMalloc((void **) &data.device_heaters, bitmap.size()));
  HANDLE_ERROR(hipMalloc((void **) &data.device_input, bitmap.size()));
  HANDLE_ERROR(hipMalloc((void **) &data.device_output, bitmap.size()));

  hipChannelFormatDesc descriptor = hipCreateChannelDesc<float>();
  
  HANDLE_ERROR(hipBindTexture2D(NULL, texture_heaters,
    data.device_heaters, descriptor, DIM, DIM, sizeof(float) * DIM));
  HANDLE_ERROR(hipBindTexture2D(NULL, texture_input,
    data.device_input, descriptor, DIM, DIM, sizeof(float) * DIM));
  HANDLE_ERROR(hipBindTexture2D(NULL, texture_output,
    data.device_output, descriptor, DIM, DIM, sizeof(float) * DIM));

  float *heaters_grid = new float[bitmap.size()];
  for (int i = 0; i < DIM * DIM; i++) {
    heaters_grid[i] = 0.0f;

    int x = i % DIM;
    int y = i / DIM;

    if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
      heaters_grid[i] = MAX_TEMPERATURE;

    heaters_grid[100 + 100 * DIM] = (MIN_TEMPERATURE + MAX_TEMPERATURE) / 2;
    heaters_grid[100 + 700 * DIM] = MIN_TEMPERATURE;
    heaters_grid[300 + 300 * DIM] = MIN_TEMPERATURE;
    heaters_grid[700 + 200 * DIM] = MIN_TEMPERATURE;

    for (int k = 800; k < 900; k++) {
      for (int j = 400; j < 500; j++)
        heaters_grid[j + k * DIM] = MIN_TEMPERATURE;
    }
  }

  HANDLE_ERROR(hipMemcpy(data.device_heaters, heaters_grid,
    bitmap.size(), hipMemcpyHostToDevice));

  for (int j = 800; j < DIM; j++)
    for (int i = 0; i < 200; i++)
      heaters_grid[i + j * DIM] = MAX_TEMPERATURE;

  HANDLE_ERROR(hipMemcpy(data.device_heaters, heaters_grid,
    bitmap.size(), hipMemcpyHostToDevice));

  delete [] heaters_grid;

  bitmap.animate((void (*)(void *, int)) animate_display,
    (void (*)(void *)) animate_exit);

  return 0;
}


void animate_display(Data *data, int clock)
{
  float compute_time;
  
  HANDLE_ERROR(hipEventRecord(data->start, 0));

  AnimateBitmap *bitmap = data->bitmap;

  dim3 blocks_per_grid(DIM / 16, DIM / 16);
  dim3 threads_per_block(16, 16);
  volatile bool flag_IO = true;

  for (int i = 0; i < TIME_STEPS_PER_FRAME; i++) {
    float *input_ptr, *output_ptr;
    
    if (flag_IO == true) {
      input_ptr = data->device_input;
      output_ptr = data->device_output;
    } else {
      input_ptr = data->device_output;
      output_ptr = data->device_input;
    }
    
    maintain_heaters<<<blocks_per_grid, threads_per_block>>>(
      input_ptr);
    temperature_update<<<blocks_per_grid, threads_per_block>>>(
      output_ptr, flag_IO);

    flag_IO = !flag_IO;
  }

  float_to_color<<<blocks_per_grid, threads_per_block>>>(
    data->device_input, data->device_bitmap);
  
  HANDLE_ERROR(hipMemcpy(bitmap->pointer(), data->device_bitmap,
    bitmap->size(), hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipEventRecord(data->stop, 0));
  HANDLE_ERROR(hipEventSynchronize(data->stop));
  HANDLE_ERROR(hipEventElapsedTime(&compute_time,
    data->start, data->stop));
  data->compute_time += compute_time;
  
  ++data->count_frames;

  printf("Mean compute time per frame: %.1f ms\n",
    data->compute_time / data->count_frames);
}


void animate_exit(Data *data)
{
  HANDLE_ERROR(hipUnbindTexture(texture_heaters));
  HANDLE_ERROR(hipUnbindTexture(texture_input));
  HANDLE_ERROR(hipUnbindTexture(texture_output));

  HANDLE_ERROR(hipFree(data->device_heaters));
  HANDLE_ERROR(hipFree(data->device_input));
  HANDLE_ERROR(hipFree(data->device_output));
  
  HANDLE_ERROR(hipEventDestroy(data->start));
  HANDLE_ERROR(hipEventDestroy(data->stop));
}


__global__ void maintain_heaters(float *ptr)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int current_idx = x + y * blockDim.x * gridDim.x;

  float heat_source = tex2D(texture_heaters, x, y);
  if (heat_source != 0)
    ptr[current_idx] = heat_source;
}


__global__ void temperature_update(float *ptr, bool flag_IO)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int current_idx = x + y * blockDim.x * gridDim.x;

  float current, left, right, upper, lower;

  if (flag_IO) {
    current = tex2D(texture_input, x, y);
    left = tex2D(texture_input, x - 1, y);
    right = tex2D(texture_input, x + 1, y);
    upper = tex2D(texture_input, x, y - 1);
    lower = tex2D(texture_input, x, y + 1);
  } else {
    current = tex2D(texture_output, x, y);
    left = tex2D(texture_output, x - 1, y);
    right = tex2D(texture_output, x + 1, y);
    upper = tex2D(texture_output, x, y - 1);
    lower = tex2D(texture_output, x, y + 1);
  }

  float gradient = left + right + upper + lower - 4 * current;
  ptr[current_idx] = current + SPEED * gradient;
}
