#include "hip/hip_runtime.h"
/* Histogram computation from an array of random bytes, using
   GPU hardware-supported global memory atomics for avoiding
   thread read and write access conflicts in the updates to
   our histogram's 256 bins.
   
   Namely, the code herewith mostly introduces the atomic sequence of operations
   encompassed by CUDA C's atomicAdd(address, value).
   
   However, the performance is poor compared to a simple CPU implementation;
   this is improved by using shared instead of global memory, as illustrated
   in histogram_with_shared_memory_atomics.cu, part of the same directory.
*/


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "../Common/errors.h"


#define BINS 256
#define SIZE ((long int) (100 * 1024 * 1024)) // 100MB of random bytes


__global__ void kernel(unsigned char *, long, unsigned int *);
unsigned char* random_bytes_stream(int);


int main(void)
{
  hipEvent_t start, stop;
  float compute_time;
  
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  srand(time(NULL));
  unsigned char *stream = random_bytes_stream(SIZE);

  unsigned char *device_stream;
  unsigned int *device_histogram;

  HANDLE_ERROR(hipMalloc((void **) &device_stream, SIZE));
  HANDLE_ERROR(hipMemcpy(device_stream, stream, SIZE,
    hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **) &device_histogram,
    sizeof(unsigned int) * BINS));
  HANDLE_ERROR(hipMemset(device_histogram, 0,
    sizeof(unsigned int) * BINS));

  hipDeviceProp_t properties;
  HANDLE_ERROR(hipGetDeviceProperties(&properties, 0));
  kernel<<<4 * properties.multiProcessorCount, BINS>>>(
    device_stream, SIZE, device_histogram);

  unsigned int *histogram = new unsigned int[BINS]();
  HANDLE_ERROR(hipMemcpy(histogram, device_histogram, sizeof(int) * BINS,
    hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&compute_time, start, stop));
  printf("Histogram computation took %.1f ms\n", compute_time);

  long total = 0;
  for (int i = 0; i < BINS; i++)
    total += histogram[i];
    
  printf("Total sum of histogram bins vs. expected: %ld vs. %ld",
    total, SIZE);

  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  
  HANDLE_ERROR(hipFree(device_stream));
  HANDLE_ERROR(hipFree(device_histogram));
  delete [] histogram;

  return 0;
}


__global__ void kernel(
  unsigned char *stream, long size, unsigned int *histogram)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  while (id < size) {
    atomicAdd(&(histogram[stream[id]]), 1);
    id += blockDim.x * gridDim.x;
  }
}


unsigned char* random_bytes_stream(int n)
{
  unsigned char *stream = (unsigned char *) malloc(n);
  HANDLE_NULL(stream);

  for (int i = 0; i < n; i++)
    stream[i] = rand();

  return stream;
}
