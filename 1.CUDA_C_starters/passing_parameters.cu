#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../Common/errors.h"


__global__ void add(int i, int j, int *k)
{
  *k = i + j;
}

int main(void)
{
  int i;
  int *device_ptr;

  HANDLE_ERROR(hipMalloc((void **) &device_ptr, sizeof(int)));
  add<<<1, 1>>>(2, 7, device_ptr);
  HANDLE_ERROR(hipMemcpy(&i, device_ptr, sizeof(int), hipMemcpyDeviceToHost));
  
  printf("2 + 7 = %d\n", i);

  hipFree(device_ptr);

  return 0;
}
